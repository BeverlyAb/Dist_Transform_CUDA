#include "hip/hip_runtime.h"
//#include <cstdio>
//#include <cstdlib>
#include <assert.h> //This is assert to check for conditions in kernels
#include <stdlib.h>
#include <stdio.h>
//#include <cmath>
#include "timer.h"
#include "cuda_utils.h"
#include "pnmfile.h"
#include "imconv.h"
#include "dt.h"
typedef float dtype;
typedef unsigned char dtype2;

#define N_ (8 * 1024 * 1024)
#define MAX_THREADS 256
#define MAX_BLOCKS 64
#define MAX_WIDTH_HEIGHT 500
#define CUDA_ERROR_CHECK
#define MIN(x,y) ((x < y) ? x : y)
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

/* return the next power of 2 number that is larger than x */
unsigned int nextPow2( unsigned int x ) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

/* find out # of threads and # thread blocks for a particular kernel */
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
  if (whichKernel < 3)
    {
      /* 1 thread per element */
      threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
      blocks = (n + threads - 1) / threads;
    }
  else
    {
      /* 1 thread per 2 elements */
      threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
      blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }
  /* limit the total number of threads */
  if (whichKernel == 5)
    blocks = MIN(maxBlocks, blocks);
}

/* special type of reduction to account for floating point error */
dtype reduce_cpu(dtype *data, int n) {
  dtype sum = data[0];
  dtype c = (dtype)0.0;
  for (int i = 1; i < n; i++)
    {
      dtype y = data[i] - c;
      dtype t = sum + y;
      c = (t - sum) - y;
      sum = t;
    }
  return sum;
}

__device__ void

dt_i(float f[], int n) 
{
    float d[MAX_WIDTH_HEIGHT];
    int v[MAX_WIDTH_HEIGHT];
    float z[MAX_WIDTH_HEIGHT+1];
    int k = 0;
    float temp_sum = 0.0;
    v[0] = 0;
    z[0] = -INF;
    z[1] = +INF;
	
    for(int q=0;q<MAX_WIDTH_HEIGHT;q++)
    {
	v[q]=0;
    }
    for (int q = 1; q <= n-1; q++) {
      float s  = ((f[q]+(q*q))-(f[v[k]]+(v[k]*v[k])))/(2*q-2*v[k]);
      while (s <= z[k]) {
        k--;
	float temp = f[q];
	
	int t = v[k];
//Below 3 for debugging
	t= (t<n)?t:n-1;
	t= (t>-1)?t:0;
	float temp2 = f[t];
	temp_sum = temp+temp2+t;	
       // s  = ((f[q]+(q*q))-(f[v[k]]+(v[k]*v[k])))/(2*q-2*v[k]);
        s  = ((f[q]+(q*q))-(f[t]+(v[k]*v[k])))/(2*q-2*v[k]);
      }
      k++;
	
      v[k] = q;
      z[k] = s;
      z[k+1] = +INF;
    }
  
    k = 0;
    for (int q = 0; q <= n-1; q++) {
      while (z[k+1] < q)
        k++;
      d[q] = (q-v[k])*(q-v[k]) + f[v[k]] + temp_sum; //!!!!!REMOVE_TEMP_SUM DEBUG - added for debugging
    }
  
    for(int q=0;q<n;q++)
    { 
      f[q] = d[q];
    }
  
  }

__global__ void
kernel_thresh (dtype *input, dtype *output, unsigned int n)
{
  __shared__  dtype scratch[MAX_THREADS];

  unsigned int bid = gridDim.x * blockIdx.y + blockIdx.x;
  unsigned int i = bid * blockDim.x + threadIdx.x;

  if(i < n) {
    scratch[threadIdx.x] = input[i]; 
  } else {
    scratch[threadIdx.x] = 0;
  }
  __syncthreads ();

    if((threadIdx.x % (3)) == 0) {
      scratch[threadIdx.x] = scratch[threadIdx.x] + 20;
    }

    output[i] = scratch[threadIdx.x];
  __syncthreads ();
}


inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

__global__ void
kernel_all_pix (dtype2 *input, dtype2 *output, unsigned int width,unsigned int height)
{
//One row stored in shared memory
//Number of blocks = height
//For now launch 1 block with height  number of threads
 __shared__  dtype2 scratch[400];

  unsigned int img_index = threadIdx.x*width;

  __syncthreads ();
	for(int j=0;j<width;j++)
	{
		if(j>20 && j<80)
		output[img_index+j]= 40;
		else
		output[img_index+j]= input[img_index+j];
	}

  __syncthreads ();


}

__global__ void
kernel_all_pix_float (dtype *input, dtype *output, unsigned int width,unsigned int height)
{
//One row stored in shared memory
//Number of blocks = height
//For now launch 1 block with height  number of threads
 //__shared__  dtype2 scratch[400];

  //unsigned int img_index = threadIdx.x*width;

    unsigned int row_num = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int img_index = (row_num)*width;

  // assert(row_num<height);
  __syncthreads ();
    if(row_num < height)
{  

    float f[MAX_WIDTH_HEIGHT];

    for (int x = 0; x < width; x++) 
    {
      f[x] = input[img_index+x];
    }
    dt_i(f, width);
  
  
  
    for (int x = 0; x < width; x++) 
    {
      output[img_index+x] = f[x];
    }

}


/*  __syncthreads ();
	for(int j=0;j<width;j++)
	{
		if(j>20 && j<80)
		output[img_index+j]= 40;
		else
		output[img_index+j]= input[img_index+j];
	}
*/
  __syncthreads ();


}


void all_pix (dtype2 *input, dtype2 *output, unsigned int width,unsigned int height)

{

for(int i=0;i<height;i++)
{
  unsigned int img_index = i*width;
	for(int j=width/2;j<width;j++)
	{
		output[img_index+j]= input[img_index+j];
//		input[img_index+j]= scratch[j];
	}

}

}


int 
main(int argc, char** argv)
{

  if (argc != 3) {
    fprintf(stderr, "usage: %s input(pbm) output(pgm)\n", argv[0]);
    return 1;
  }
  char *input_name = argv[1];
  char *output_name = argv[2];
  image<uchar> *input = loadPGM(input_name);
//------------Basic DT ------//
 image<float> *out = dt(input);
  int height = input-> height();
  int width = input->width();
for (int y = 0; y < out->height(); y++) {
    for (int x = 0; x < out->width(); x++) {
      imRef(out, x, y) = sqrt(imRef(out, x, y));
    }
  }
  image<uchar> *gray = imageFLOATtoUCHAR(out);
//-----------------------------//
  int N = width*height;
/*  
  dtype2 *h_idata, *h_odata, h_cpu;
 
  dtype2 *d_idata, *d_odata;	

*/


  dtype *h_idata, *h_odata, h_cpu;
  dtype *d_idata, *d_odata;	

  image<dtype> *input_float = imageUCHARtoFLOAT(input);
  image<dtype> *output_img = new image<dtype>(width, height, false);

  


  h_idata = (dtype*) malloc (N * sizeof (dtype));
  h_odata = (dtype*) malloc (N * sizeof (dtype));
  CUDA_CHECK_ERROR (hipMalloc (&d_idata,N * sizeof (dtype)));
  CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * sizeof (dtype)));



/* //Switch to this in case of dtype2
  h_idata = (dtype2*) malloc (N * sizeof (dtype2));
  h_odata = (dtype2*) malloc (N * sizeof (dtype2));
  CUDA_CHECK_ERROR (hipMalloc (&d_idata,N * sizeof (dtype2)));
  CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * sizeof (dtype2)));
*/
  h_idata = input_float->data;

  dim3 gb(1,1, 1);
  dim3 tb(height, 1, 1);

  

  //CUDA_CHECK_ERROR (hipMemcpy (d_idata,h_idata, N * sizeof (dtype2), 
//				hipMemcpyHostToDevice));

  CUDA_CHECK_ERROR (hipMemcpy (d_idata,h_idata, N * sizeof (dtype), 
				hipMemcpyHostToDevice));


  kernel_all_pix_float <<<gb, tb>>> (d_idata, d_odata, width,height);
  hipDeviceSynchronize ();

  kernel_all_pix_float <<<gb, tb>>> (d_idata, d_odata,width,height);
  CudaCheckError();

  hipDeviceSynchronize ();

  //CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N* sizeof (dtype2), hipMemcpyDeviceToHost));
  CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N* sizeof (dtype), hipMemcpyDeviceToHost));
 

 image<dtype> *transpose_img = new image<dtype>(height, width, false); //Note: Here height, width oppositve of above, doesn't matter though because memory allocated same

  /*----Below loop is to do transpose, make this part parallel later*/
   for(int i=0;i<height;i++)
  {
    for(int j=0;j<width;j++)
    {
      transpose_img->data[j*width+i] = h_odata[i*width+j];  

    }
  }

  dim3 gb2(2,1, 1);
  dim3 tb2(width, 1, 1);

  h_idata = transpose_img->data;



  CUDA_CHECK_ERROR (hipMemcpy (d_idata,h_idata, N * sizeof (dtype), 
        hipMemcpyHostToDevice));


  kernel_all_pix_float <<<gb2, tb2>>> (d_idata, d_odata, height,width); //reversed width,height
  hipDeviceSynchronize ();

   


  kernel_all_pix_float <<<gb2, tb2>>> (d_idata, d_odata,height,width); //reversed width,height
  CudaCheckError();

  hipDeviceSynchronize ();



  //CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N* sizeof (dtype2), hipMemcpyDeviceToHost));
  CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N* sizeof (dtype), hipMemcpyDeviceToHost));
  
  
//This section is to do the tranpose again
   for(int i=0;i<height;i++)
  {
    for(int j=0;j<width;j++)
    {
      output_img->data[i*width+j] = h_odata[j*width+i];
    }
  }

  
  //image<uchar> *out_res= imageFLOATtoUCHAR(output_img,0.0,255.0);
    image<uchar> *out_res = new image<uchar>(width,height,false); 
  for(int i=0;i<height;i++)
  {
  	for(int j=0;j<width;j++)
  	{
  		out_res->data[i*width+j] = (uchar)(2.276 * sqrt(output_img->data[i*width+j]));		//Hardcoding scale value here, need to find min ,max automatically and do it properly
  	}
  }



  savePGM(out_res, output_name);




 

/*===================================================*/


/*===================================================*/
  int tN = 256;
  dtype *th_idata, *th_odata, th_cpu;
  dtype *td_idata, *td_odata;	

  th_idata = (dtype*) malloc (tN * sizeof (dtype));
  th_odata = (dtype*) malloc (tN * sizeof (dtype));
  CUDA_CHECK_ERROR (hipMalloc (&td_idata,tN * sizeof (dtype)));
  CUDA_CHECK_ERROR (hipMalloc (&td_odata, tN * sizeof (dtype)));
  for(int i = 0; i < tN; i++) {
    th_idata[i] = i;

	}

  dim3 tgb(1,1, 1);
  dim3 ttb(tN, 1, 1);

  /* warm up */
  

  CUDA_CHECK_ERROR (hipMemcpy (td_idata,th_idata, tN * sizeof (dtype), 
				hipMemcpyHostToDevice));



  kernel_thresh <<<tgb, ttb>>> (td_idata, td_odata, tN);//To warm up
  hipDeviceSynchronize ();

  kernel_thresh <<<tgb, ttb>>> (td_idata, td_odata,tN);
  hipDeviceSynchronize ();

  CUDA_CHECK_ERROR (hipMemcpy (th_odata, td_odata, tN* sizeof (dtype), hipMemcpyDeviceToHost));
  
for(int i=0;i<20;i++)
	{
		printf("%d=%0.1f ",i,th_odata[i]);
	}
printf("\n");


/*===================================================*/

  return 0;
}
