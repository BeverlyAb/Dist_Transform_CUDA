#include "hip/hip_runtime.h"
//#include <cstdio>
//#include <cstdlib>
#include <assert.h> //This is assert to check for conditions in kernels
#include <stdlib.h>
#include <stdio.h>
//#include <cmath>
#include "timer.h"
#include "cuda_utils.h"
#include "pnmfile.h"
#include "imconv.h"
#include "dt.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
typedef float dtype;
typedef unsigned char dtype2;

#define N_ (8 * 1024 * 1024)
#define MAX_THREADS 256
#define MAX_BLOCKS 64
#define MAX_WIDTH_HEIGHT 500
#define CUDA_ERROR_CHECK
#define MIN(x,y) ((x < y) ? x : y)
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

/* return the next power of 2 number that is larger than x */
unsigned int nextPow2( unsigned int x ) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

/* find out # of threads and # thread blocks for a particular kernel */
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
  if (whichKernel < 3)
    {
      /* 1 thread per element */
      threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
      blocks = (n + threads - 1) / threads;
    }
  else
    {
      /* 1 thread per 2 elements */
      threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
      blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }
  /* limit the total number of threads */
  if (whichKernel == 5)
    blocks = MIN(maxBlocks, blocks);
}

/* special type of reduction to account for floating point error */
dtype reduce_cpu(dtype *data, int n) {
  dtype sum = data[0];
  dtype c = (dtype)0.0;
  for (int i = 1; i < n; i++)
    {
      dtype y = data[i] - c;
      dtype t = sum + y;
      c = (t - sum) - y;
      sum = t;
    }
  return sum;
}

__device__ void

dt_i(float f[], int n) 
{
    float d[MAX_WIDTH_HEIGHT];
    int v[MAX_WIDTH_HEIGHT];
    float z[MAX_WIDTH_HEIGHT+1];
    int k = 0;
    float temp_sum = 0.0;
    v[0] = 0;
    z[0] = -INF;
    z[1] = +INF;
	
    for(int q=0;q<MAX_WIDTH_HEIGHT;q++)
    {
	v[q]=0;
    }
    for (int q = 1; q <= n-1; q++) {
      float s  = ((f[q]+(q*q))-(f[v[k]]+(v[k]*v[k])))/(2*q-2*v[k]);
      while (s <= z[k]) {
        k--;
	
        s  = ((f[q]+(q*q))-(f[v[k]]+(v[k]*v[k])))/(2*q-2*v[k]);
       // s  = ((f[q]+(q*q))-(f[t]+(v[k]*v[k])))/(2*q-2*v[k]);
      }
      k++;
	
      v[k] = q;
      z[k] = s;
      z[k+1] = +INF;
    }
  
    k = 0;
    for (int q = 0; q <= n-1; q++) {
      while (z[k+1] < q)
        k++;
      d[q] = (q-v[k])*(q-v[k]) + f[v[k]] ; //!!!!!REMOVE_TEMP_SUM DEBUG - added for debugging
    }
  
    for(int q=0;q<n;q++)
    { 
      f[q] = d[q];
    }
  
  }

__global__ void
kernel_thresh (dtype *input, dtype *output, unsigned int n)
{
  __shared__  dtype scratch[MAX_THREADS];

  unsigned int bid = gridDim.x * blockIdx.y + blockIdx.x;
  unsigned int i = bid * blockDim.x + threadIdx.x;

  if(i < n) {
    scratch[threadIdx.x] = input[i]; 
  } else {
    scratch[threadIdx.x] = 0;
  }
  __syncthreads ();

    if((threadIdx.x % (3)) == 0) {
      scratch[threadIdx.x] = scratch[threadIdx.x] + 20;
    }

    output[i] = scratch[threadIdx.x];
  __syncthreads ();
}


inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

__global__ void
kernel_all_pix (dtype2 *input, dtype2 *output, unsigned int width,unsigned int height)
{
//One row stored in shared memory
//Number of blocks = height
//For now launch 1 block with height  number of threads
 __shared__  dtype2 scratch[400];

  unsigned int img_index = threadIdx.x*width;

  __syncthreads ();
	for(int j=0;j<width;j++)
	{
		if(j>20 && j<80)
		output[img_index+j]= 40;
		else
		output[img_index+j]= input[img_index+j];
	}

  __syncthreads ();


}

__global__ void
kernel_all_pix_float (dtype *input, dtype *output, unsigned int width,unsigned int height)
{
//One row stored in shared memory
//Number of blocks = height
//For now launch 1 block with height  number of threads
 //__shared__  dtype2 scratch[400];

  //unsigned int img_index = threadIdx.x*width;

    unsigned int row_num = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int img_index = (row_num)*width;

  // assert(row_num<height);
  __syncthreads ();

    float f[MAX_WIDTH_HEIGHT];

    for (int x = 0; x < width; x++) 
    {
      f[x] = input[img_index+x];
    }
    dt_i(f, width);
  
  
  
    for (int x = 0; x < width; x++) 
    {
      output[img_index+x] = f[x];
    }



/*	  __syncthreads ();
	for(int j=0;j<width;j++)
	{
		if(j>20 && j<80)
		output[img_index+j]= 40;
		else
		output[img_index+j]= input[img_index+j];
	}
*/
  	__syncthreads ();


}


void all_pix (dtype2 *input, dtype2 *output, unsigned int width,unsigned int height)

{

for(int i=0;i<height;i++)
{
  unsigned int img_index = i*width;
	for(int j=width/2;j<width;j++)
	{
		output[img_index+j]= input[img_index+j];
//		input[img_index+j]= scratch[j];
	}

}

}


int 
main(int argc, char** argv)
{

  if (argc != 3) {
    fprintf(stderr, "usage: %s input(pbm) output(pgm)\n", argv[0]);
    return 1;
  }
  char *input_name = argv[1];
  char *output_name = argv[2];
  image<uchar> *input = loadPGM(input_name);
//------------Basic DT ------//
 image<float> *out = dt(input);
  int height = input-> height();
  int width = input->width();
for (int y = 0; y < out->height(); y++) {
    for (int x = 0; x < out->width(); x++) {
     // imRef(out, x, y) = sqrt(imRef(out, x, y));
    }
  }
  image<uchar> *gray = imageFLOATtoUCHAR(out);
//-----------------------------//
  int N = width*height;
/*  
  dtype2 *h_idata, *h_odata, h_cpu;
 
  dtype2 *d_idata, *d_odata;	

*/


  dtype *h_idata, *h_odata, h_cpu;
  dtype *d_idata, *d_odata;	

  image<dtype> *input_float = imageUCHARtoFLOAT(input);
  image<dtype> *output_img = new image<dtype>(width, height, false);

  


  h_idata = (dtype*) malloc (N * sizeof (dtype));
  h_odata = (dtype*) malloc (N * sizeof (dtype));
  CUDA_CHECK_ERROR (hipMalloc (&d_idata,N * sizeof (dtype)));
  CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * sizeof (dtype)));



/* //Switch to this in case of dtype2
  h_idata = (dtype2*) malloc (N * sizeof (dtype2));
  h_odata = (dtype2*) malloc (N * sizeof (dtype2));
  CUDA_CHECK_ERROR (hipMalloc (&d_idata,N * sizeof (dtype2)));
  CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * sizeof (dtype2)));
*/
  h_idata = input_float->data;

  dim3 gb(1,1, 1);
  dim3 tb(height, 1, 1);

  

  //CUDA_CHECK_ERROR (hipMemcpy (d_idata,h_idata, N * sizeof (dtype2), 
//				hipMemcpyHostToDevice));

  CUDA_CHECK_ERROR (hipMemcpy (d_idata,h_idata, N * sizeof (dtype), 
				hipMemcpyHostToDevice));


  kernel_all_pix_float <<<gb, tb>>> (d_idata, d_odata, width,height);
  hipDeviceSynchronize ();

  kernel_all_pix_float <<<gb, tb>>> (d_idata, d_odata,width,height);
  CudaCheckError();

  hipDeviceSynchronize ();

  //CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N* sizeof (dtype2), hipMemcpyDeviceToHost));
  CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N* sizeof (dtype), hipMemcpyDeviceToHost));
 

 image<dtype> *transpose_img = new image<dtype>(height, width, false); //Note: Here height, width oppositve of above, doesn't matter though because memory allocated same

  /*----Below loop is to do transpose, make this part parallel later*/
   for(int i=0;i<width;i++)
  {
    for(int j=0;j<height;j++)
    {
      transpose_img->data[i*height+j] = h_odata[j*width+i];  

    }
  }

/*
 for(int i=0;i<width;i++)
  {
    for(int j=0;j<height;j++)
    {
      gray_trans->data[i*height + j] = input->data[j*width + i];
    }
  }

*/

  dim3 gb2(2,1, 1);
  dim3 tb2(200, 1, 1);

  dtype *hidata2;
  hidata2 = transpose_img->data;

  dtype *hodata2;
  hodata2 = transpose_img->data;


  CUDA_CHECK_ERROR (hipMemcpy (d_idata,hidata2, N * sizeof (dtype), 
        hipMemcpyHostToDevice));


  kernel_all_pix_float <<<gb2, tb2>>> (d_idata, d_odata, height,width); //reversed width,height
  hipDeviceSynchronize ();

   


  kernel_all_pix_float <<<gb2, tb2>>> (d_idata, d_odata,height,width); //reversed width,height
  CudaCheckError();

  hipDeviceSynchronize ();



  //CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N* sizeof (dtype2), hipMemcpyDeviceToHost));
  CUDA_CHECK_ERROR (hipMemcpy (hodata2, d_odata, N* sizeof (dtype), hipMemcpyDeviceToHost));
  
  
//This section is to do the tranpose again
   for(int i=0;i<width;i++)
  {
    for(int j=0;j<height;j++)
    {
      output_img->data[j*width+i] = hodata2[i*height+j];
    }
  }

  
  //image<uchar> *out_res= imageFLOATtoUCHAR(output_img,0.0,255.0);
    image<uchar> *out_res = new image<uchar>(width,height,false);
    //output_img->data = out->data; 
    float min_val = output_img->data[0];
    float max_val = min_val;
for(int i=0;i<height;i++)
{
	for(int j=0;j<width;j++)
	{
		if(max_val < output_img->data[i*width+j]) max_val = output_img->data[i*width+j];
		if(min_val > output_img->data[i*width+j]) min_val = output_img->data[i*width+j];
	
	}
}   

/*
 for(int i=0;i<height;i++)
{
	for(int j=0;j<width;j++)
	{
		if(max_val < out->data[i*width+j]) max_val = out->data[i*width+j];
		if(min_val > out->data[i*width+j]) min_val = out->data[i*width+j];
	
	}
}
*/
	float scale = 255/(sqrt(max_val)-sqrt(min_val));
	printf("max:%0.2f min:%0.2f s=%0.2f\n",max_val,min_val,scale);	
  for(int i=0;i<height;i++)
  {
  	for(int j=0;j<width;j++)
  	{
  		out_res->data[i*width+j] = (uchar)(scale * (sqrt(output_img->data[i*width+j])-sqrt(min_val)));		//Hardcoding scale value here, need to find min ,max automatically and do it properly
  		//out_res->data[i*width+j] = (uchar)(output_img->data[i*width+j]);		//Hardcoding scale value here, need to find min ,max automatically and do it properly
  	}
  }



  savePGM(out_res, output_name);




 

/*===================================================*/


/*===================================================*/
  int tN = 256;
  dtype *th_idata, *th_odata, th_cpu;
  dtype *td_idata, *td_odata;	

  th_idata = (dtype*) malloc (tN * sizeof (dtype));
  th_odata = (dtype*) malloc (tN * sizeof (dtype));
  CUDA_CHECK_ERROR (hipMalloc (&td_idata,tN * sizeof (dtype)));
  CUDA_CHECK_ERROR (hipMalloc (&td_odata, tN * sizeof (dtype)));
  for(int i = 0; i < tN; i++) {
    th_idata[i] = i;

	}

  dim3 tgb(1,1, 1);
  dim3 ttb(tN, 1, 1);

  /* warm up */
  

  CUDA_CHECK_ERROR (hipMemcpy (td_idata,th_idata, tN * sizeof (dtype), 
				hipMemcpyHostToDevice));



  kernel_thresh <<<tgb, ttb>>> (td_idata, td_odata, tN);//To warm up
  hipDeviceSynchronize ();

  kernel_thresh <<<tgb, ttb>>> (td_idata, td_odata,tN);
  hipDeviceSynchronize ();

  CUDA_CHECK_ERROR (hipMemcpy (th_odata, td_odata, tN* sizeof (dtype), hipMemcpyDeviceToHost));
  
for(int i=0;i<20;i++)
	{
		printf("%d=%0.1f ",i,th_odata[i]);
	}
printf("\n");


/*===================================================*/
/*
thrust::host_vector<int> h_vec( 16*1024*1024 );
thrust::generate(h_vec.begin(), h_vec.end(), rand);
// transfer data to the device
thrust::device_vector<int> d_vec = h_vec;
thrust::sort(d_vec.begin(), d_vec.end()); // sort data on the device
// transfer data back to host
thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
printf("thrust finished sorting\n");
*/
  return 0;
}
